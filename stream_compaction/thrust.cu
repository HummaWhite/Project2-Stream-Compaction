#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::host_vector<int> in(idata, idata + n);
            thrust::device_vector<int> devIn = in;
            thrust::device_vector<int> devOut(n);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

            auto last = thrust::exclusive_scan(devIn.begin(), devIn.end(), devOut.begin());
            
            timer().endGpuTimer();

            int size = last - devOut.begin();
            thrust::host_vector<int> out = devOut;
            memcpy(odata, out.data(), size * sizeof(int));
        }

        void sort32u(uint32_t* out, uint32_t* in, uint32_t n) {
            thrust::host_vector<uint32_t> array(in, in + n);
            thrust::device_vector<uint32_t> devArray = array;

            timer().startGpuTimer();
           
            thrust::sort(devArray.begin(), devArray.end());

            timer().endGpuTimer();

            array = devArray;
            memcpy(out, array.data(), n * sizeof(uint32_t));
        }

        void stableSort32u(uint32_t* out, uint32_t* in, uint32_t n) {
            thrust::host_vector<uint32_t> array(in, in + n);
            thrust::device_vector<uint32_t> devArray = array;

            timer().startGpuTimer();

            thrust::stable_sort(devArray.begin(), devArray.end());

            timer().endGpuTimer();

            array = devArray;
            memcpy(out, array.data(), n * sizeof(uint32_t));
        }
    }
}
